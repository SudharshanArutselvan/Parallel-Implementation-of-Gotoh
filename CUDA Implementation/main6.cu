#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gotoh.h"
#include "inline.cc"
#include<iostream>
#include "timer.h"
using namespace std;

__device__ int maximum(int arg0, int arg1, int arg2, int *idx)
{

    if (arg0 >= arg1 && arg0 >= arg2) {
        *idx = 0;
        return arg0;
    }
    if (arg1 >= arg0 && arg1 >= arg2) {
       *idx = 1;
        return arg1;
    }

    *idx = 2;
    return arg2;
}

__global__ void block_max(int *a, int no,int N, int *b, int *gapX, int *gapY, const int n)

{
    __shared__ int mat[3][3][3];
    int index_i=0,index_j=0;

    if(no<N/n)
    {
        index_i=n*blockIdx.x;
        index_j=n*no-index_i;
    }
    else
    {
        index_i=(blockIdx.x+no-N/n+1)*n;
        index_j=n*no-index_i;
    }

    int index = index_j*(N+1)*3+index_i*3;

    //column by column copying; per column, 3 rows (tid.x), each tid.x has 3 tid.y 
    for(int i=0;i<n+1;i++)
    {    
        mat[threadIdx.x][i][threadIdx.y]=a[index+i*(N+1)*3+3*threadIdx.x+threadIdx.y];
    }

    for(int i=0;i<2*n-1;i++)
    {
        if(i<n)
        {
            if(threadIdx.x<=i)
            {
                if(threadIdx.y==0)
                    mat[threadIdx.x+1][i-threadIdx.x+1][0]+=maximum(mat[threadIdx.x][i-threadIdx.x+1][0], mat[threadIdx.x][i-threadIdx.x+1][1] + gapY[index_j+1], mat[threadIdx.x][i-threadIdx.x+1][2] + gapY[index_j+1], &b[index+(i-threadIdx.x+1)*(N+1)*3+(threadIdx.x+1)*3]);

                if(threadIdx.y==1) 
                    mat[threadIdx.x+1][i-threadIdx.x+1][1]+=maximum(mat[threadIdx.x+1][i-threadIdx.x][0] + gapX[index_i+1], mat[threadIdx.x+1][i-threadIdx.x][1], mat[threadIdx.x+1][i-threadIdx.x][2] + gapX[index_i+1], &b[index+(i-threadIdx.x+1)*(N+1)*3+(threadIdx.x+1)*3+1]);

                if(threadIdx.x==2)
                    mat[threadIdx.x+1][i-threadIdx.x+1][2]+=maximum(mat[threadIdx.x][i-threadIdx.x][0], mat[threadIdx.x][i-threadIdx.x][1], mat[threadIdx.x][i-threadIdx.x][2], &b[index+(i-threadIdx.x+1)*(N+1)*3+(threadIdx.x+1)*3+2]);
            }
        }
        else
        {
            if(threadIdx.x<i-n+1)
            {
                if(threadIdx.y==0)
                    mat[n-threadIdx.x][i+2-n+threadIdx.x][0]+=maximum(mat[n-threadIdx.x-1][i+2-n+threadIdx.x][0], mat[n-threadIdx.x-1][i+2-n+threadIdx.x][1] + gapY[index_j+1], mat[n-threadIdx.x-1][i+2-n+threadIdx.x][2] + gapY[index_j+1], &b[index+(i+2-n+threadIdx.x)*(N+1)*3+(n-threadIdx.x)*3]);

                if(threadIdx.y==1)
                     mat[n-threadIdx.x][i+2-n+threadIdx.x][1]+=maximum(mat[n-threadIdx.x][i+2-n+threadIdx.x-1][0] + gapX[index_i+1], mat[n-threadIdx.x][i+2-n+threadIdx.x-1][1], mat[n-threadIdx.x][i+2-n+threadIdx.x-1][2] + gapX[index_i+1], &b[index+(i+2-n+threadIdx.x)*(N+1)*3+(n-threadIdx.x)*3+1]);

                if(threadIdx.y==2)
                     mat[n-threadIdx.x][i+2-n+threadIdx.x][2]+=maximum(mat[n-threadIdx.x-1][i+2-n+threadIdx.x-1][0] , mat[n-threadIdx.x-1][i+2-n+threadIdx.x-1][1], mat[n-threadIdx.x-1][i+2-n+threadIdx.x-1][2], &b[index+(i+2-n+threadIdx.x)*(N+1)*3+(n-threadIdx.x)*3+2]);     
            }
        }
    }
 
__syncthreads();

    for(int j=0;j<n+1;j++)
    {
        a[index+3*(N+1)*j+3*threadIdx.x+threadIdx.y]=mat[threadIdx.x][j][threadIdx.y];
    }
}

void recur(gth_Arr arr)
{
//assuming two strings of equal length
int N=arr.lenX;
int size=(N+1)*(N+1)*3*sizeof(int); 

//Creating reduntant copies of data, copying to 1D array explicitly
int *data_copy=(int*)malloc(size);
int *path_copy=(int*)malloc(size);
for(int i=0;i<=arr.lenX;i++)
{
    for(int j=0;j<=arr.lenY;j++){
        for(int k=0;k<3;k++)
        {    data_copy[j*(arr.lenX+1)*3+i*3+k]=arr.data[i][j][k];
             path_copy[j*(arr.lenX+1)*3+i*3+k]=arr.path[i][j][k];
        }
       }  
}

int *gapX=(int*)malloc((N+1)*sizeof(int));
int *gapY=(int*)malloc((N+1)*sizeof(int));

for(int i=0;i<N+1;i++)
 {
    gapX[i]=arr.gapX[i];
    gapY[i]=arr.gapY[i];
}

//Creating variables for device memory
 int *dev_a, *dev_b, *dev_gapX, *dev_gapY;
 int n=2;

    hipMalloc(&dev_a, size);
    hipMalloc(&dev_b, size);
    hipMalloc(&dev_gapX, (N+1)*sizeof(int));
    hipMalloc(&dev_gapY, (N+1)*sizeof(int));

    hipMemcpy(dev_a,data_copy,size,hipMemcpyHostToDevice);
    hipMemcpy(dev_b, path_copy,size,  hipMemcpyHostToDevice);
    hipMemcpy(dev_gapX, gapX, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_gapY, gapY, (N+1)*sizeof(int), hipMemcpyHostToDevice);

    dim3 thread_count(n+1,3);

    struct stopwatch_t* timer=NULL;
    long double t_gpu;
    stopwatch_init();

    timer=stopwatch_create();
    stopwatch_start(timer);
    for(int i=0;i<(2*N-1)/n;i++)
    {
        if(i<N/2)
        {
            block_max<<<i+1,thread_count>>>(dev_a,i,N,dev_b,dev_gapX,dev_gapY,n);
        }
       else
        {
            block_max<<<(2*N-1)/n-i-N%n,thread_count>>>(dev_a,i,N,dev_b,dev_gapX,dev_gapY,n);
        }
    }
    t_gpu=stopwatch_stop(timer);
    cout<<"GPU: "<<t_gpu;
    hipMemcpy(data_copy,dev_a,size,hipMemcpyDeviceToHost);
    hipMemcpy(path_copy,dev_b,size,hipMemcpyDeviceToHost);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_gapX);
    hipFree(dev_gapY);

for(int i=0;i<=N;i++)
{
    for(int j=0;j<=N;j++){
        for(int k=0;k<3;k++)
        {   arr.data[i][j][k]= data_copy[j*(arr.lenX+1)*3+i*3+k];
            arr.path[i][j][k]=path_copy[j*(arr.lenX+1)*3+i*3+k];
        }
       }
}

if(N%2!=0)
{
	for(int j=1;j<N;j++)
	{
		arr.data[N][j][X] += maximum2(
                arr.data[N-1][j  ][X],
                arr.data[N-1][j  ][Y] + arr.gapY[j],
                arr.data[N-1][j  ][Z] + arr.gapY[j],
            &arr.path[N][j][X]);

            arr.data[N][j][Y] += maximum2(
                arr.data[N  ][j-1][X] + arr.gapX[N],
                arr.data[N  ][j-1][Y],
                arr.data[N  ][j-1][Z] + arr.gapX[N],
            &arr.path[N][j][Y]);

            arr.data[N][j][Z] += maximum2(
                arr.data[N-1][j-1][X],
                arr.data[N-1][j-1][Y],
                arr.data[N-1][j-1][Z],
            &arr.path[N][j][Z]);

	    arr.data[j][N][X] += maximum2(
                arr.data[j-1][N  ][X],
                arr.data[j-1][N  ][Y] + arr.gapY[N],
                arr.data[j-1][N  ][Z] + arr.gapY[N],
            &arr.path[j][N][X]);

            arr.data[j][N][Y] += maximum2(
                arr.data[j  ][N-1][X] + arr.gapX[j],
                arr.data[j  ][N-1][Y],
                arr.data[j  ][N-1][Z] + arr.gapX[j],
            &arr.path[j][N][Y]);

            arr.data[j][N][Z] += maximum2(
                arr.data[j-1][N-1][X],
                arr.data[j-1][N-1][Y],
                arr.data[j-1][N-1][Z],
            &arr.path[j][N][Z]);

	}
	int i=N,j=N;
	arr.data[i][j][X] += maximum2(
                arr.data[i-1][j  ][X],
                arr.data[i-1][j  ][Y] + arr.gapY[j],
                arr.data[i-1][j  ][Z] + arr.gapY[j],
            &arr.path[i][j][X]);

            arr.data[i][j][Y] += maximum2(
                arr.data[i  ][j-1][X] + arr.gapX[i],
                arr.data[i  ][j-1][Y],
                arr.data[i  ][j-1][Z] + arr.gapX[i],
            &arr.path[i][j][Y]);

            arr.data[i][j][Z] += maximum2(
                arr.data[i-1][j-1][X],
                arr.data[i-1][j-1][Y],
                arr.data[i-1][j-1][Z],
            &arr.path[i][j][Z]);
}

}

int gth_align(gth_Arr arr) {

    for (size_t i=1 ; i<=arr.lenX ; i++) {
        arr.gapX[i] = arr.data[i][0][Y];
        arr.data[i][0][X] += arr.data[i-1][0][X];
    }

    for (size_t j=1 ; j<=arr.lenY ; j++) {
        arr.gapY[j] = arr.data[0][j][X];
        arr.data[0][j][Y] += arr.data[0][j-1][Y];
    }

    fill_arr(X, arr.path, X, 1, arr.lenX, 0,        0);
    fill_arr(Y, arr.path, Y, 0,        0, 1, arr.lenY);

    fill_arr(-Inf, arr.data, Y, 1, arr.lenX, 0,        0);
    fill_arr(-Inf, arr.data, Z, 1, arr.lenX, 0,        0);
    fill_arr(-Inf, arr.data, X, 0,        0, 1, arr.lenY);
    fill_arr(-Inf, arr.data, Z, 0,        0, 1, arr.lenY);

    recur(arr);

    for (size_t i=0 ; i<=arr.lenX ; i++) arr.gapX[i] = 0;
    for (size_t j=0 ; j<=arr.lenY ; j++) arr.gapY[j] = 0;

    // backtracking
    int K;
    int max = maximum2(
        arr.data[arr.lenX][arr.lenY][X],
        arr.data[arr.lenX][arr.lenY][Y],
        arr.data[arr.lenX][arr.lenY][Z],
    &K);

    size_t i = arr.lenX, j = arr.lenY;
    while (i > 0 || j > 0) {
        if (K == X) {
            arr.gapY[j]++;
            K = arr.path[i--][j  ][X];
        }
        else if (K == Y) {
            arr.gapX[i]++;
            K = arr.path[i  ][j--][Y];
        }
        else if (K == Z) {
            K = arr.path[i--][j--][Z];
        }

    }

    return max;
}

// default BLOSUM62 matrix from the EMBOSS package. See end of the file for its initialization.
//const gth_Sub BLOSUM62;
const gth_Sub BLOSUM62 = {
    .alpha = "ARNDCQEGHILKMFPSTWYVBZX", .score = 
    {{  4, -2,  0, -2, -1, -2,  0, -2, -1, -4, -1, -1, -1, -2, -4, -1, -1, -1,  1,  0, -4,  0, -3,  0, -2, -1},
     { -2,  4, -3,  4,  1, -3, -1,  0, -3, -4,  0, -4, -3,  3, -4, -2,  0, -1,  0, -1, -4, -3, -4, -1, -3,  1},
     {  0, -3,  9, -3, -4, -2, -3, -3, -1, -4, -3, -1, -1, -3, -4, -3, -3, -3, -1, -1, -4, -1, -2, -2, -2, -3},
     { -2,  4, -3,  6,  2, -3, -1, -1, -3, -4, -1, -4, -3,  1, -4, -1,  0, -2,  0, -1, -4, -3, -4, -1, -3,  1},
     { -1,  1, -4,  2,  5, -3, -2,  0, -3, -4,  1, -3, -2,  0, -4, -1,  2,  0,  0, -1, -4, -2, -3, -1, -2,  4},
     { -2, -3, -2, -3, -3,  6, -3, -1,  0, -4, -3,  0,  0, -3, -4, -4, -3, -3, -2, -2, -4, -1,  1, -1,  3, -3},
     {  0, -1, -3, -1, -2, -3,  6, -2, -4, -4, -2, -4, -3,  0, -4, -2, -2, -2,  0, -2, -4, -3, -2, -1, -3, -2},
     { -2,  0, -3, -1,  0, -1, -2,  8, -3, -4, -1, -3, -2,  1, -4, -2,  0,  0, -1, -2, -4, -3, -2, -1,  2,  0},
     { -1, -3, -1, -3, -3,  0, -4, -3,  4, -4, -3,  2,  1, -3, -4, -3, -3, -3, -2, -1, -4,  3, -3, -1, -1, -3},
     { -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4},
     { -1,  0, -3, -1,  1, -3, -2, -1, -3, -4,  5, -2, -1,  0, -4, -1,  1,  2,  0, -1, -4, -2, -3, -1, -2,  1},
     { -1, -4, -1, -4, -3,  0, -4, -3,  2, -4, -2,  4,  2, -3, -4, -3, -2, -2, -2, -1, -4,  1, -2, -1, -1, -3},
     { -1, -3, -1, -3, -2,  0, -3, -2,  1, -4, -1,  2,  5, -2, -4, -2,  0, -1, -1, -1, -4,  1, -1, -1, -1, -1},
     { -2,  3, -3,  1,  0, -3,  0,  1, -3, -4,  0, -3, -2,  6, -4, -2,  0,  0,  1,  0, -4, -3, -4, -1, -2,  0},
     { -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4},
     { -1, -2, -3, -1, -1, -4, -2, -2, -3, -4, -1, -3, -2, -2, -4,  7, -1, -2, -1, -1, -4, -2, -4, -2, -3, -1},
     { -1,  0, -3,  0,  2, -3, -2,  0, -3, -4,  1, -2,  0,  0, -4, -1,  5,  1,  0, -1, -4, -2, -2, -1, -1,  3},
     { -1, -1, -3, -2,  0, -3, -2,  0, -3, -4,  2, -2, -1,  0, -4, -2,  1,  5, -1, -1, -4, -3, -3, -1, -2,  0},
     {  1,  0, -1,  0,  0, -2,  0, -1, -2, -4,  0, -2, -1,  1, -4, -1,  0, -1,  4,  1, -4, -2, -3,  0, -2,  0},
     {  0, -1, -1, -1, -1, -2, -2, -2, -1, -4, -1, -1, -1,  0, -4, -1, -1, -1,  1,  5, -4,  0, -2,  0, -2, -1},
     { -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4},
     {  0, -3, -1, -3, -2, -1, -3, -3,  3, -4, -2,  1,  1, -3, -4, -2, -2, -3, -2,  0, -4,  4, -3, -1, -1, -2},
     { -3, -4, -2, -4, -3,  1, -2, -2, -3, -4, -3, -2, -1, -4, -4, -4, -2, -3, -3, -2, -4, -3, 11, -2,  2, -3},
     {  0, -1, -2, -1, -1, -1, -1, -1, -1, -4, -1, -1, -1, -1, -4, -2, -1, -1,  0,  0, -4, -1, -2, -1, -1, -1},
     { -2, -3, -2, -3, -2,  3, -3,  2, -1, -4, -2, -1, -1, -2, -4, -3, -1, -2, -2, -2, -4, -1,  2, -1,  7, -2},
     { -1,  1, -3,  1,  4, -3, -2,  0, -3, -4,  1, -3, -1,  0, -4, -1,  3,  0,  0, -1, -4, -2, -3, -1, -2,  4}}
};


int main(int argc, char **argv) {

    // arguments parsing
    
    int error = 0;
    int dump = 0;
    int quiet = 0;
    char *seqX_path = NULL;
    char *seqY_path = NULL;
    char *matrix_path = NULL;
    char *arr_paths[] = {NULL, NULL, NULL};
    double gapopen = 9.5;
    double gapextend = 0.5;
    double endopen = 0.0;
    double endextend = 0.0;

    for (size_t i=1 ; i<argc && !error ; i++) {
        if (!strcmp(argv[i], "-dump")) dump = 1;
        else if (!strcmp(argv[i], "-quiet")) quiet = 1;
        else if (argv[i][0] == '-') {
            if (i+1 >= argc) {
                fprintf(stderr, "ERROR: missing argument for option '%s'\n\n", argv[i]);
                error = 1;
            }
            else if (!strcmp(argv[i], "-matrix"))    matrix_path  = argv[i+1];
            else if (!strcmp(argv[i], "-gapopen"))   gapopen      = atof(argv[i+1]);
            else if (!strcmp(argv[i], "-gapextend")) gapextend    = atof(argv[i+1]);
            else if (!strcmp(argv[i], "-endopen"))   endopen      = atof(argv[i+1]);
            else if (!strcmp(argv[i], "-endextend")) endextend    = atof(argv[i+1]);
            else if (!strcmp(argv[i], "-arrxfile"))  arr_paths[0] = argv[i+1];
            else if (!strcmp(argv[i], "-arryfile"))  arr_paths[1] = argv[i+1];
            else if (!strcmp(argv[i], "-arrzfile"))  arr_paths[2] = argv[i+1];
            else {
                fprintf(stderr, "ERROR: unknown option '%s'\n\n", argv[i]);
                error = 1;
            }
            i++;
        }
        else if (seqX_path == NULL) seqX_path = argv[i];
        else if (seqY_path == NULL) seqY_path = argv[i];
        else {
            fprintf(stderr, "ERROR: too many arguments\n\n");
            error = 1;
        }
    }

    if (argc <= 1) error = 1;
    else if ((seqX_path == NULL || seqY_path == NULL) && !error) {
        fprintf(stderr, "ERROR: you must provide two sequence files\n\n");
        error = 1;
    }


    // help message

    if (error) {
        printf("Needleman-Wunsch global alignment of two sequences\n\n");
        printf("Usage: %s [OPTIONS] SEQUENCE1.fasta SEQUENCE2.fasta\n\n", argv[0]);
        printf("Options and their defaults:\n");
        printf("    -matrix     BLOSUM62    matrix file in NCBI/EMBOSS format\n");
        printf("    -gapopen    9.5         opening gap penalty\n");
        printf("    -gapextend  0.5         extending gap penalty\n");
        printf("    -endopen    0.0         opening end gap penalty\n");
        printf("    -endextend  0.0         extending end gap penalty\n");
        printf("    -quiet                  decrease verbosity\n\n");
        printf("Advanced:\n");
        printf("    -arrxfile   arrx.txt    load initial array X from this file\n");
        printf("    -arryfile   arry.txt    load initial array Y from this file\n");
        printf("    -arrzfile   arrz.txt    load initial array Z from this file\n");
        printf("    -dump                   output final arrays containing partial scores\n\n");
        printf("Default parameters are the same as those of EMBOSS Needle.\n");
        printf("Feeding corrupted file formats to the program may result in undefined behaviour\n");
        return 0;
    }


    // load sequences

    gth_Seq seqX = gth_read_fasta(seqX_path);
    if (seqX.len == 0) {
        fprintf(stderr, "ERROR: problem reading file '%s'\n", seqX_path);
        return -1;
    }

    gth_Seq seqY = gth_read_fasta(seqY_path);
    if (seqY.len == 0) {
        fprintf(stderr, "ERROR: problem reading file '%s'\n", seqY_path);
        return -1;
    }


    // load matrix

    gth_Sub matrix = BLOSUM62;
    if (matrix_path != NULL) {
        matrix = gth_read_matrix(matrix_path);
        if (matrix.alpha[0] == '\0') {
            fprintf(stderr, "ERROR: problem reading file '%s'\n", matrix_path);
            return -1;
        }
    }
    for (size_t i=0 ; i<26 ; i++) {
        for (size_t j=0 ; j<26 ; j++)
        {    matrix.score[i][j] *= 10;
        }
    }

    // create, fill, and backtrack the arrays
    gth_Arr array = gth_init(seqX.len, seqY.len);
    gth_set_sub(array, seqX.res, seqY.res, matrix.score);

    gth_set_gap(array, (int)(gapopen*10), (int)(gapextend*10), (int)(endopen*10), (int)(endextend*10));
    for (int k=0 ; k<3 ; k++) {
        if (arr_paths[k] == NULL) continue;
        FILE *file = fopen(arr_paths[k], "r");
        if (!file) {
            fprintf(stderr, "ERROR: problem reading file '%s'\n", arr_paths[k]);
            gth_free(array);
            free(seqX.res);
            free(seqY.res);
            return -1;
        }
        for (size_t i=0 ; i<=array.lenX ; i++) {
            for (size_t j=0 ; j<=array.lenY ; j++) {
                fscanf(file, "%d", &array.data[i][j][k]);
                array.data[i][j][k] *= 10;
            }
            fscanf(file, "%*[^\n]");
        }
        fclose(file);
    }
    double score = (double)(gth_align(array)) / 10;


    // output

    if (!quiet) {
        printf("# Needleman-Wunsch global alignment of two sequences\n");
        printf("#\n");
        if (arr_paths[2] == NULL) {
            printf("# matrix file: %s\n", (matrix_path == NULL) ? "none specified, using BLOSUM62" : matrix_path);
        }
        else {
            printf("# substitution scores array provided by user\n");
        }
        if (arr_paths[0] == NULL && arr_paths[1] == NULL) {
            printf("# gap opening penalty: %.1f\n", gapopen);
            printf("# gap extending penalty: %.1f\n", gapextend);
            printf("# end gap opening penalty: %.1f\n", endopen);
            printf("# end gap extending penalty: %.1f\n", endextend);
        }
        else {
            printf("# gap scores array(s) provided by user\n");
        }
        printf("#\n");
        printf("# score: %.1f\n\n", score);
    }
    if (dump) {
        for (int k=0 ; k<3 ; k++) {
            printf("# scores in array %c:\n", 'X'+k);
            for (size_t i=0 ; i<=array.lenX ; i++) {
                for (size_t j=0 ; j<=array.lenY ; j++) {
                    printf("% 15d ", array.data[i][j][k]);
                }
                printf("\n");
            }
            printf("\n");
        }
    }
    printf(">%s\n", seqX.name);
    gth_putseq(stdout, seqX.res, array.gapX);
    printf("\n\n>%s\n", seqY.name);
    gth_putseq(stdout, seqY.res, array.gapY);
    printf("\n");


    // cleanup

    gth_free(array);
    free(seqX.res);
    free(seqY.res);
    return 0;
}


// default BLOSUM62 matrix from the EMBOSS package.

